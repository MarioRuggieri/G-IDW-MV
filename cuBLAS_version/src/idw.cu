#include "hip/hip_runtime.h"
#include "idw.h"

void checkCUDAError(const char* msg)
{
    hipError_t err = hipGetLastError();
    
    if (err != hipSuccess)
    {
        fprintf(stderr, "Cuda error: %s %s\n", msg, hipGetErrorString(err));
        exit(-1);
    }
}

__device__ float havesineDistGPU(Point2D p1, Point2D p2)
{
    float   lat1 = PI*p1.y/180,
            lat2 = PI*p2.y/180,
            dlat = PI*(p2.y-p1.y)/180,
            dlon = PI*(p2.x-p1.x)/180,
            a = sin(dlat/2) * sin(dlat/2) + cos(lat1) * cos(lat2) * sin(dlon/2) * sin(dlon/2),
            c = 2 * atan2(sqrt(a), sqrt(1-a));

    return R * c;
}

__global__ void divideByWsum(float *zValues, float *wSum, int QN)
{
    int ind = threadIdx.x + blockIdx.x*blockDim.x;

    if (ind < QN) 
        zValues[ind] /= wSum[ind];
}

// IDW parallel GPU version
__global__ void computeWeights(     Point2D *knownPoints, 
                                    Point2D *queryPoints, 
                                    float *W, 
                                    int KN, 
                                    int QN, 
                                    int stride,
                                    float *wSum,
                                    int nIter,
                                    int MAX_SHMEM_SIZE)
{
    extern __shared__ Point2D shMem[];
    int ind = threadIdx.x + blockIdx.x*blockDim.x, smStartInd, startInd, i, k, currentKN, shift, work = 1;
    float my_wSum = 0, w, d;
    Point2D myPoint, p;
    
    shift = 0;
    currentKN = MAX_SHMEM_SIZE;	//chunk current dimension

    // each iteration fills as much as possible shared memory
    for (k = 0; k < nIter; k++)
    {
        //the last or only one iteration
        if (currentKN > KN) currentKN = KN;
        
        /* --- loading known points into shared memory --- */
        
        smStartInd = threadIdx.x*stride;

        //shift used to move into knownPoints array for chunk selection
        startInd = smStartInd + shift;  
        //if (ind == 0) printf("startInd: %d\n",startInd);

        if (startInd < currentKN) 
        {
            i = 0;
            while (i < stride && (startInd + i) < currentKN) // for the last thread: <= stride points
            {
                shMem[smStartInd + i] = knownPoints[startInd + i];
                i++;
            }
        }

        __syncthreads();

        /* --- loading finished --- */
        
        if (work)
        {
            // updating the interpolated z value for each thread
            if (ind < QN) 
            {
                myPoint = queryPoints[ind]; // some block threads are not used

                for (i = 0; i < currentKN-shift; i++)
                {
                    p = shMem[i];

                    d = havesineDistGPU(myPoint,p);
                    //d = sqrt((myPoint.x - p.x)*(myPoint.x - p.x) + (myPoint.y - p.y)*(myPoint.y - p.y));

                    if (d != 0)
                    {
                        //if (d < SEARCH_RADIUS)
                        //{
                        //if (ind == 0) printf("%d\n",ind*KN + i+k*MAX_SHMEM_SIZE);
                            w = 1/(d*d);
                            W[ind*KN + i+k*MAX_SHMEM_SIZE] = w;
                            my_wSum += w;
                        /*}
                        else
                        {
                            W[ind*KN + i+k*MAX_SHMEM_SIZE] = 0;
                        }*/
                    }
                    else
                    {
                        for (int l=0; l<KN; l++) W[ind*KN + l] = 0;
                        W[ind*KN + i+k*MAX_SHMEM_SIZE] = 1; //1 for the zero distance point
                        my_wSum = 1;
                        work = 0;
                        break; 
                    }
                }
            }       
        }
	    
	shift = currentKN;
        currentKN += MAX_SHMEM_SIZE; 

        __syncthreads();
    }

    if (ind < QN)
    {
        wSum[ind] = my_wSum;
    }
    
}

float havesineDistCPU(Point2D p1, Point2D p2)
{
    float   lat1 = PI*p1.y/180,
            lat2 = PI*p2.y/180,
            dlat = PI*(p2.y-p1.y)/180,
            dlon = PI*(p2.x-p1.x)/180,
            a = sin(dlat/2) * sin(dlat/2) + cos(lat1) * cos(lat2) * sin(dlon/2) * sin(dlon/2),
            c = 2 * atan2(sqrt(a), sqrt(1-a));

    return R * c;
}

// IDW sequential CPU version
void sequentialIDW(Point2D *knownPoints, float* knownValues, Point2D *queryPoints, float *zValues, int KN, int QN)
{
    int i,j;
    float wSum, w, d;
    
    for (i=0; i<QN; i++)
    {
        wSum = 0; zValues[i] = 0;

        for (j=0; j<KN; j++)
        {
            d = havesineDistCPU(queryPoints[i],knownPoints[j]);
            /*d = sqrt( (queryPoints[i].x - knownPoints[j].x)*(queryPoints[i].x - knownPoints[j].x) + 
                        (queryPoints[i].y - knownPoints[j].y)*(queryPoints[i].y - knownPoints[j].y));*/

            if (d != 0)
            {
                //if (d < SEARCH_RADIUS)
                //{
                    w = 1/(d*d);
                    wSum += w;
                    zValues[i] += w*knownValues[j];
                //}
            }
            else
            {
                zValues[i] = knownValues[j];
		        wSum = 1;
                break;
            }
        }
        
        zValues[i] /= wSum;
    }
}

// Random generation of 2D known points and 2D query points
void generateRandomData(Point2D *knownPoints, float *knownValues, Point2D *queryPoints, int KN, int QN)
{
    int i;
    srand((unsigned int)time(NULL));

    for (i=0; i<KN; i++)
    {
        knownPoints[i].x = (rand()/(float)(RAND_MAX))* 180;
        knownPoints[i].y = (rand()/(float)(RAND_MAX))* 180;
        knownValues[i] = (rand()/(float)(RAND_MAX))* 180;
    }

    for (i=0; i<QN; i++)
    {
        queryPoints[i].x = (rand()/(float)(RAND_MAX))* 90;
        queryPoints[i].y = (rand()/(float)(RAND_MAX))* 90;
    }
    
}

int getLines(char *filename)
{
    FILE *fp = fopen(filename,"r");
    int ch = 0, cont = 1;

    while(!feof(fp))
    {
        ch = fgetc(fp);
        if(ch == '\n')
        {
            cont++;
        }
    }

    fclose(fp);

    return cont;
}


void generateDataset(char *filename, Point2D *knownLocations, float *knownValues)
{
    FILE *fp = fopen(filename,"r");
    int i=0;

    while(fscanf(fp,"%f;%f;%f;",&knownLocations[i].x,&knownLocations[i].y,&knownValues[i]) == 3)
    {
        i++;
    }

    fclose(fp);
}

void generateGrid(char *filename, Point2D *queryLocations)
{
    FILE *fp = fopen(filename,"r");
    int i=0;

    while(fscanf(fp,"%f;%f;",&queryLocations[i].x,&queryLocations[i].y) == 2 )
    {
        i++;
    }

    fclose(fp);
}


int saveData(Point2D *knownPoints, int KN, Point2D *queryPoints, float *zValues, float *zValuesGPU, int QN, float cpuElapsedTime, float gpuElaspedTime)
{
    FILE *f;
    time_t t;
    struct tm *tm;
    char *directory, date[30], *myDir;

    t = time(NULL);
    tm = localtime(&t);
    strftime(date, sizeof(date)-1, "%d-%m-%Y_%H:%M:%S", tm);
    directory = "Results-";

    myDir = (char *)malloc(strlen(directory)+strlen(date)+1);
    strcpy(myDir, directory);
    strcat(myDir, date);

    if( mkdir(myDir,0777) < 0 ) 
    {
       printf("Cannot create directory\n");
       return(-1);   
    }

    if (chdir(myDir) < 0)
    {
        printf("Cannot change directory\n");
        return(-1);
    }

    // Saving generated data
    /*
    f = fopen("generatedData.txt", "w");
    if (f == NULL)
    {
        printf("Error opening generatedData file!\n");
        return(-1);
    }
    
    for (int i=0; i<KN; i++)
        fprintf(f, "(x: %f, y: %f, z: %f)\n", knownPoints[i].x, knownPoints[i].y, knownPoints[i].z);
    
    fclose(f);
    */
    // Saving CPU output
    f = fopen("cpuOutput.txt", "w");
    if (f == NULL)
    {
        printf("Error opening cpuOutput file!\n");
        return(-1);
    }
    
    for (int i=0; i<QN; i++)
        fprintf(f, "(x: %f, y: %f, z: %f)\n", queryPoints[i].x, queryPoints[i].y, zValues[i]);
    
    fclose(f);

    // Saving GPU output
    f = fopen("gpuOutput.txt", "w");
    if (f == NULL)
    {
        printf("Error opening gpuOutput file!\n");
        return(-1);
    }
    
    for (int i=0; i<QN; i++)
        fprintf(f, "(x: %f, y: %f, z: %f)\n", queryPoints[i].x, queryPoints[i].y, zValuesGPU[i]);
    
    fclose(f);

    // Saving times
    f = fopen("times.txt", "w");
    if (f == NULL)
    {
        printf("Error opening times file!\n");
        return(-1);
    }
    
    fprintf(f, "Cpu Elapsed Time: %f\n Gpu Elasped Time: %f\n Speed Up: %f", 
                cpuElapsedTime, gpuElaspedTime, cpuElapsedTime/gpuElaspedTime);
    
    fclose(f);

    return 0;
}

int updateLog(float gpuMeanTime, int QN, int KN, int nBlocks, int nThreadsForBlock)
{
    FILE *f;

    f = fopen("log.txt","a");
    if (f == NULL)
    {
        printf("Error opening log!\n");
        return(-1);
    }

    fprintf(f, "KnownPointsNum: %d QueryPointsNum: %d BlockNum: %d ThreadNumForBlock: %d Time: %f s\n", 
                KN, QN, nBlocks, nThreadsForBlock, gpuMeanTime);
    
    fclose(f);

    return 0;
}

int updateLogCpuGpu(float gpuMeanTime, float cpuMeanTime, float gpuSTD, float cpuSTD, int QN, int KN, int nBlocks, int nThreadsForBlock)
{
    FILE *f;

    f = fopen("fullLog.txt","a");
    if (f == NULL)
    {
        printf("Error opening log!\n");
        return(-1);
    }

    fprintf(f, "KnownPointsNum: %d QueryPointsNum: %d BlockNum: %d ThreadNumForBlock: %d CPUMeanTime: %f s CPUstd: %f GPUMeanTime: %f s GPUstd: %f\n", 
                KN, QN, nBlocks, nThreadsForBlock, cpuMeanTime, cpuSTD, gpuMeanTime, gpuSTD);
    
    fclose(f);

    return 0;
}

void getMaxAbsError(float *zValues, float *zValuesGPU, int QN, float *maxErr)
{
    int i;
    float err;

    *maxErr = -1;

    for (i = 0; i < QN; i++)
    {
        err = abs(zValues[i]-zValuesGPU[i]);
        if (err > 1e-6)
            printf("ERRORE\n%lf ; %lf\n\n",zValues[i],zValuesGPU[i]);
        if (err > *maxErr)
            *maxErr = err;
    }
}

float getRes(float *ref, float *result, int QN)
{
    int i;
    float res = 0, ref_norm = 0;

    for (i = 0; i < QN; i++)
    {
        ref_norm += ref[i]*ref[i];
    }

    ref_norm = sqrt(ref_norm);

    for (i = 0; i < QN; i++)
    {
        res += (ref[i]-result[i])*(ref[i]-result[i]);
    }

    return sqrt(res)/ref_norm;
}

float getSTD(float xm, float x[], int N)
{
    float s = 0;
    for (int i=0; i<N; i++)
    {
        s += pow(x[i] - xm,2);
    }

    s /= N-1;

    return sqrt(s);
}

